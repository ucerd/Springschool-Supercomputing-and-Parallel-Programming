
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 1000
#define BLOCK_SIZE 16

__global__ void matrixMultiplyKernel(int *a, int *b, int *result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        result[row * n + col] = sum;
    }
}

void printMatrix(int *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d\t", mat[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int *mat1, *mat2, *result;
    int size = N * N * sizeof(int);

    // Allocate memory for matrices on the host
    mat1 = (int *)malloc(size);
    mat2 = (int *)malloc(size);
    result = (int *)malloc(size);
    if (mat1 == NULL || mat2 == NULL || result == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        return EXIT_FAILURE;
    }

    // Initialize matrices with random values
    for (int i = 0; i < N * N; i++) {
        mat1[i] = rand() % 1000; // Random values between 0 and 999
        mat2[i] = rand() % 1000; // Random values between 0 and 999
    }

    // Allocate memory for matrices on the device
    int *d_mat1, *d_mat2, *d_result;
    hipMalloc((void **)&d_mat1, size);
    hipMalloc((void **)&d_mat2, size);
    hipMalloc((void **)&d_result, size);

    // Copy matrices from host to device
    hipMemcpy(d_mat1, mat1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, mat2, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    struct timeval start, end;
    gettimeofday(&start, NULL); // Start timing

    // Launch kernel
    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_mat1, d_mat2, d_result, N);

    // Copy result matrix from device to host
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    gettimeofday(&end, NULL); // End timing
    double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

    // Print a subset of the matrices for demonstration
    printf("Matrix 1 (subset):\n");
    printMatrix(mat1, 5, 5);
    printf("\nMatrix 2 (subset):\n");
    printMatrix(mat2, 5, 5);
    printf("\nResultant Matrix (subset):\n");
    printMatrix(result, 5, 5);
    printf("\nExecution time: %lf seconds\n", elapsed);

    // Free allocated memory on the device
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_result);

    // Free allocated memory on the host
    free(mat1);
    free(mat2);
    free(result);

    return 0;
}

